/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>


// Number of particles
int N;
__device__ int hN;

//  Lennard-Jones parameters in natural units!
double sigma = 1.;
double epsilon = 1.;
double m = 1.;
double kB = 1.;

double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)

//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;
//  Vectors!
//
const int MAXPART=5001;
//  Position
double r[MAXPART][3];
//  Velocity
double v[MAXPART][3];
//  Acceleration
double a[MAXPART][3];
//  Force
double F[MAXPART][3];

// atom type
char atype[10];
//  Function prototypes
//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  
//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double* VelocityVerlet(double dt, int iter, FILE *fp);  
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
double computeAccelerationsPotential();
//  Numerical Recipes function for generation gaussian distribution
double gaussdist();
//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();
//  Compute mean squared velocity from particle velocities
double MeanSquaredVelocity();
//  Compute total kinetic energy from particle mass and velocities
double Kinetic();

int main()
{   
    //  variable delcarations
    int i;
    double dt, Vol, Temp, Press, Pavg, Tavg, rho;
    double VolFac, TempFac, PressFac, timefac;
    double KE, PE, mvs, gc, Z;
    char prefix[1000], tfn[1000], ofn[1000], afn[1000];
    FILE *tfp, *ofp, *afp;
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  WELCOME TO WILLY P CHEM MD!\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  ENTER A TITLE FOR YOUR CALCULATION!\n");
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  TITLE ENTERED AS '%s'\n",prefix);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    /*     Table of values for Argon relating natural units to SI units:
     *     These are derived from Lennard-Jones parameters from the article
     *     "Liquid argon: Monte carlo and molecular dynamics calculations"
     *     J.A. Barker , R.A. Fisher & R.O. Watts
     *     Mol. Phys., Vol. 21, 657-673 (1971)
     *
     *     mass:     6.633e-26 kg          = one natural unit of mass for argon, by definition
     *     energy:   1.96183e-21 J      = one natural unit of energy for argon, directly from L-J parameters
     *     length:   3.3605e-10  m         = one natural unit of length for argon, directly from L-J parameters
     *     volume:   3.79499-29 m^3        = one natural unit of volume for argon, by length^3
     *     time:     1.951e-12 s           = one natural unit of time for argon, by length*sqrt(mass/energy)
     ***************************************************************************************/
    
    //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //  Edit these factors to be computed in terms of basic properties in natural units of
    //  the gas being simulated
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("  WHICH NOBLE GAS WOULD YOU LIKE TO SIMULATE? (DEFAULT IS ARGON)\n");
    printf("\n  FOR HELIUM,  TYPE 'He' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR NEON,    TYPE 'Ne' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR ARGON,   TYPE 'Ar' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR KRYPTON, TYPE 'Kr' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR XENON,   TYPE 'Xe' THEN PRESS 'return' TO CONTINUE\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        //strcpy(atype,"Ar");
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n                     YOU ARE SIMULATING %s GAS! \n",atype);
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  YOU WILL NOW ENTER A FEW SIMULATION PARAMETERS\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n\n  ENTER THE INTIAL TEMPERATURE OF YOUR GAS IN KELVIN\n");
    scanf("%lf",&Tinit);
    // Make sure temperature is a positive number!
    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }
    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
    
    
    printf("\n\n  ENTER THE NUMBER DENSITY IN moles/m^3\n");
    printf("  FOR REFERENCE, NUMBER DENSITY OF AN IDEAL GAS AT STP IS ABOUT 40 moles/m^3\n");
    printf("  NUMBER DENSITY OF LIQUID ARGON AT 1 ATM AND 87 K IS ABOUT 35000 moles/m^3\n");
    
    scanf("%lf",&rho);
    
    N = 5000;
    hipMemcpyToSymbol(HIP_SYMBOL(hN), &N, sizeof(int));
    Vol = N/(rho*NA);
    
    Vol /= VolFac;
    
    //  Limiting N to MAXPART for practical reasons
    if (N>=MAXPART) {
        
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
        
    }
    //  Check to see if the volume makes sense - is it too small?
    //  Remember VDW radius of the particles is 1 natural unit of length
    //  and volume = L*L*L, so if V = N*L*L*L = N, then all the particles
    //  will be initialized with an interparticle separation equal to 2xVDW radius
    if (Vol<N) {
        
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }
    // Vol = L*L*L;
    // Length of the box in natural units:
    L = pow(Vol,(1./3));
    
    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");     //  The MD trajectory, coordinates of every particle at each timestep
    ofp = fopen(ofn,"w");     //  Output of other quantities (T, P, gc, etc) at every timestep
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    int NumTime;
    if (strcmp(atype,"He")==0) {
        
        // dt in natural units of time s.t. in SI it is 5 f.s. for all other gasses
        dt = 0.2e-14/timefac;
        //  We will run the simulation for NumTime timesteps.
        //  The total time will be NumTime*dt in natural units
        //  And NumTime*dt multiplied by the appropriate conversion factor for time in seconds
        NumTime=50000;
    }
    else {
        dt = 0.5e-14/timefac;
        NumTime=200;
        
    }
    
    //  Put all the atoms in simple crystal lattice and give them random velocities
    //  that corresponds to the initial temperature we have specified
    initialize();
    
    //  Based on their positions, calculate the ininial intermolecular forces
    //  The accellerations of each particle will be defined from the forces and their
    //  mass, and this will allow us to update their positions via Newton's law
    computeAccelerationsPotential();
    
    
    // Print number of particles to the trajectory file
    fprintf(tfp,"%i\n",N);
    
    //  We want to calculate the average Temperature and Pressure for the simulation
    //  The variables need to be set to zero initially
    Pavg = 0;
    Tavg = 0;
    
    
    int tenp = floor(NumTime/10);
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");
    printf("  PERCENTAGE OF CALCULATION COMPLETE:\n  [");
    for (i=0; i<NumTime+1; i++) {
        
        //  This just prints updates on progress of the calculation for the users convenience
        if (i==tenp) printf(" 10 |");
        else if (i==2*tenp) printf(" 20 |");
        else if (i==3*tenp) printf(" 30 |");
        else if (i==4*tenp) printf(" 40 |");
        else if (i==5*tenp) printf(" 50 |");
        else if (i==6*tenp) printf(" 60 |");
        else if (i==7*tenp) printf(" 70 |");
        else if (i==8*tenp) printf(" 80 |");
        else if (i==9*tenp) printf(" 90 |");
        else if (i==10*tenp) printf(" 100 ]\n");
        fflush(stdout);
        
        
        // This updates the positions and velocities using Newton's Laws
        // Also computes the Pressure as the sum of momentum changes from wall collisions / timestep
        // which is a Kinetic Theory of gasses concept of Pressure
        double* result = VelocityVerlet(dt, i+1, tfp);
        Press = result[0];
        Press *= PressFac;
        
        //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        //  Now we would like to calculate somethings about the system:
        //  Instantaneous mean velocity squared, Temperature, Pressure
        //  Potential, and Kinetic Energy
        //  We would also like to use the IGL to try to see if we can extract the gas constant
        KE = Kinetic();
        mvs = KE / (N * 0.5);
        PE = result[1];
        
        // Temperature from Kinetic Theory
        Temp = m*mvs/(3*kB) * TempFac;
        
        // Instantaneous gas constant and compressibility - not well defined because
        // pressure may be zero in some instances because there will be zero wall collisions,
        // pressure may be very high in some instances because there will be a number of collisions
        gc = NA*Press*(Vol*VolFac)/(N*Temp);
        Z  = Press*(Vol*VolFac)/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;
        
        fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
        
        
    }
    
    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*(Vol*VolFac)/(N*kBSI*Tavg);
    gc = NA*Pavg*(Vol*VolFac)/(N*Tavg);
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.4e  %15.5f       %15.5f     %10.5f       %10.5f        %10.5e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  TO ANIMATE YOUR SIMULATION, OPEN THE FILE \n  '%s' WITH VMD AFTER THE SIMULATION COMPLETES\n",tfn);
    printf("\n  TO ANALYZE INSTANTANEOUS DATA ABOUT YOUR MOLECULE, OPEN THE FILE \n  '%s' WITH YOUR FAVORITE TEXT EDITOR OR IMPORT THE DATA INTO EXCEL\n",ofn);
    printf("\n  THE FOLLOWING THERMODYNAMIC AVERAGES WILL BE COMPUTED AND WRITTEN TO THE FILE  \n  '%s':\n",afn);
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.5f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.5f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.5f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.5f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.5f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.5e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    
    
    
    fclose(tfp);
    fclose(ofp);
    fclose(afp);
    
    return 0;
}


void initialize() {
    int n, p, i, j, k;
    double pos;
    
    // Number of atoms in each direction
    n = int(ceil(pow(N, 1.0/3)));
    
    //  spacing between atoms along a given direction
    pos = L / n;
    
    //  index for number of particles assigned positions
    p = 0;
    //  initialize positions
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            for (k=0; k<n; k++) {
                if (p<N) {
                    
                    r[p][0] = (i + 0.5)*pos;
                    r[p][1] = (j + 0.5)*pos;
                    r[p][2] = (k + 0.5)*pos;
                }
                p++;
            }
        }
    }
    
    // Call function to initialize velocities
    initializeVelocities();
    
}


//  Function to calculate the kinetic energy of the system
double Kinetic() { //Write Function here!  
    
    double v2, kin;
    
    kin =0.;

    for (int i=0; i<N; i++) {
            
        v2 = v[i][0]*v[i][0] + v[i][1]*v[i][1] + v[i][2]*v[i][2];
    
        kin += m*v2*0.5;
    }
    
    return kin;
    
}


// Device function to run the computations in the kernel
__global__ void computeAccelerationsPotentialGPU(double (*ak)[3], double (*rk)[3], double* Pot) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double sharedRk[128][3];

    if (i < hN) {

        for (int k = 0; k < 3; ++k)
            sharedRk[threadIdx.x][k] = rk[i][k];
        
        Pot[i] = 0;

        double rSqd, rij[3], val, vall, f, vals[3];
        double vPot_local = 0.0;
        double ak_local[3] = {0.0, 0.0, 0.0};

        for (int j = 0; j < hN; j++) {

            if (i != j) {
            
                rSqd = 0;

                rij[0] = sharedRk[threadIdx.x][0] - rk[j][0];
                rij[1] = sharedRk[threadIdx.x][1] - rk[j][1];
                rij[2] = sharedRk[threadIdx.x][2] - rk[j][2];

                rSqd = rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2];

                val = rSqd * rSqd * rSqd;
                vall = 1 / (val * val * rSqd);

                vPot_local += 4 * (rSqd - (val * rSqd)) * vall;
                
                f = (48 - 24 * val) * vall;

                vals[0] = rij[0] * f; vals[1] = rij[1] * f; vals[2] = rij[2] * f;

                ak_local[0] += vals[0];
                ak_local[1] += vals[1];
                ak_local[2] += vals[2];

            }
        }

        Pot[i] = vPot_local;
        ak[i][0] = ak_local[0];
        ak[i][1] = ak_local[1];
        ak[i][2] = ak_local[2];

    }
}


// Host function
double computeAccelerationsPotential() {
    double v_Pot[N];
    int tpb = 128;

    for (int i = 0; i < N; i++)
        a[i][0] = a[i][1] = a[i][2] = 0;

    double Pot = 0.0;

    // pointers to the device memory
    double (*ak)[3];
    double (*rk)[3];
    double* Pot_dev;

    // declare variable with size of the array in bytes
    int bytes = N * 3 * sizeof(double);

    // allocate the memory on the device
    hipMalloc((void**)&ak, bytes);
    hipMalloc((void**)&rk, bytes);
    hipMalloc((void**)&Pot_dev, N * sizeof(double));

    // copy inputs to the device
    hipMemcpy(ak, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(rk, r, bytes, hipMemcpyHostToDevice);

    int bpg = (N + tpb - 1) / tpb;
    
    computeAccelerationsPotentialGPU<<<bpg, tpb>>>(ak, rk, Pot_dev);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "Error in CUDA kernel: %s\n", hipGetErrorString(error));
    }

    // copy the output to the host (if needed)
    hipMemcpy(a, ak, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(v_Pot, Pot_dev, N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        Pot += v_Pot[i];

    // free the device memory
    hipFree(ak);
    hipFree(rk);
    hipFree(Pot_dev);

    return Pot;
}


// returns sum of dv/dt*m/A (aka Pressure) from elastic collisions with walls
double* VelocityVerlet(double dt, int iter, FILE *fp) {
    int i, j;
    double v0, v1, v2;
    double h_dt = dt*0.5;
    
    double psum = 0.;
    double* result = (double*)malloc(2*sizeof(double));
    //  Compute accelerations from forces at current position
    // this call was removed (commented) for predagogical reasons
    //computeAccelerations();
    //  Update positions and velocity with current velocity and acceleration
    for (i=0; i<N; i++) {
        //loop unrolling
        v0 = a[i][0]*h_dt;
        v1 = a[i][1]*h_dt;
        v2 = a[i][2]*h_dt;

        r[i][0] += v[i][0]*dt + v0*dt;
        r[i][1] += v[i][1]*dt + v1*dt;
        r[i][2] += v[i][2]*dt + v2*dt;
        
        v[i][0] += v0;
        v[i][1] += v1;
        v[i][2] += v2;
    }
    //  Update accellerations from updated positions
    result[1] = computeAccelerationsPotential();
    
    //  Update velocity with updated acceleration
    for (i=0; i<N; i++) {
        //loop unrolling
        v[i][0] += a[i][0]*h_dt;
        v[i][1] += a[i][1]*h_dt;
        v[i][2] += a[i][2]*h_dt;
    }
    
    // Elastic walls
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            if (r[i][j]<0.) {
                v[i][j] *=-1.; //- elastic walls
                psum += m*fabs(v[i][j]);  // contribution to pressure from "left" walls
            }
            if (r[i][j]>=L) {
                v[i][j]*=-1.;  //- elastic walls
                psum += m*fabs(v[i][j]);  // contribution to pressure from "right" walls
            }
        }
    }
    
    result[0] = psum/(6*L*L*h_dt);
    return result;
}


void initializeVelocities() {
    
    int i, j;
    
    for (i=0; i<N; i++) {
        
        for (j=0; j<3; j++) {
            //  Pull a number from a Gaussian Distribution
            v[i][j] = gaussdist();
            
        }
    }
    
    // Vcm = sum_i^N  m*v_i/  sum_i^N  M
    // Compute center-of-mas velocity according to the formula above
    double vCM[3] = {0, 0, 0};
    
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            vCM[j] += m*v[i][j];
            
        }
    }
    
    
    for (i=0; i<3; i++) vCM[i] /= N*m;
    
    //  Subtract out the center-of-mass velocity from the
    //  velocity of each particle... effectively set the
    //  center of mass velocity to zero so that the system does
    //  not drift in space!
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            v[i][j] -= vCM[j];
            
        }
    }
    
    //  Now we want to scale the average velocity of the system
    //  by a factor which is consistent with our initial temperature, Tinit
    double vSqdSum, lambda;
    vSqdSum=0.;
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            vSqdSum += v[i][j]*v[i][j];
            
        }
    }
    
    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            v[i][j] *= lambda;
            
        }
    }
}


//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}